#include "hip/hip_runtime.h"
"""
Given a matrix A and a vector b, compute the product Ab.
"""
#include<iostream>
#include<hip/hip_runtime.h>
#include<chrono>

#define cudaErrorCheck(ans){
    gpuAssert((ans),__FILE__,__LINE__);
}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
    if (code!=hipSuccess){
        fprintf(stderr,"GPUAssert: %s %s %d\n", hipGetErrorString(code),file,line);
        if(abort){
            exit(code);
        }

    }
}
__global__ void matrix_vector_multiplication_kernel(const float *A, const float *B,
    float *C, int M, int N) {

        int row  = blockIdx.y*blockDim.y + threadIdx.y;
        float cval = 0;
        if (row < M){
        for(int i=0;i<N;i++){
            cval += A[row*N+i]*B[i]; 
        }
    }
        C[row]= cval;
}


extern "C" void solve(const float *A, const float *B, float *C, int M, int N){
    dim3 threadsPerBlock(1,16);
    dim3 blocksPerGrid(1,(M+threadsPerBlock.y-1)/threadsPerBlock.y);
    matrix_vector_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M,
        N);

}

int main(){
    int M = 4;
    int N = 4;
    printf("Performing Matrix multiplication Matrix A (%d,%d) with Vector B (%d,)",M,N,N);

    //Setup the Data on the host (CPU)
    size_t size_A = (size_t)M*N*sizeof(float):
    size_t size_B = (size_t)N*sizeof(float);
    size_t size_C = (size_t)M*sizeof(float);


    // Create empty Matrices with some Data(e.g, random numbers)
    float *h_A = new float[M*N];
    float *h_B = new float[N];
    float *h_C = new float[M];

    for(int i=0; i<M*N; ++i){
        h_A[i] = (float)rand() / RAND_MAX;
    }
    for(int i=0; i<N; ++i){
        h_B[i] = (float)rand() / RAND_MAX;
    }

    //Allocate memory on device
    float *d_A, *d_B, *d_C;
    cudaErrorCheck(hipMalloc(&d_A, size_A));
    cudaErrorCheck(hipMalloc(&d_B, size_B));
    cudaErrorCheck(hipMalloc(&d_C, size_C));

    //Copy data from host to device
    printf("Copying Data from Host to Device");
    cudaErrorCheck(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
    
    // Call solve
    solve(d_A,d_B,d_C,M,N);

    cudaErrorCheck(hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost));

    for(int i =0; i<M, ++i){
        printf("%d",h_C[i])
    }


}